
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <unistd.h>
#include <string>

#define THREADS_PER_BLOCK 16
uint secondsToSleep = 1;

__global__ void arrayDifference(const float *a, const float *b, float *results, size_t elementCount)
{
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < elementCount)
    {
        results[i] = a[i] - b[i];
    }
}

__host__ void executeKernel(float *a, float *b, float *results, size_t elementCount)
{
    dim3 dimGrid((elementCount + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1, 1);
    dim3 dimBlock(THREADS_PER_BLOCK, 1, 1);
    arrayDifference<<<dimGrid, dimBlock>>>(a, b, results, elementCount);
    hipDeviceSynchronize();
    std::cout << "Input files loaded and processed, results:\n";
    for (size_t i = 0; i < elementCount; i++)
    {
        std::cout << results[i] << (i == elementCount - 1 ? "\n" : ",");
    }
}

__host__ void loadInputFile(float *a, float *b, size_t elementCount)
{
    bool inputIsReady = false;
    while (!inputIsReady)
    {
        std::ifstream lockA("./input_a.lock");
        std::ifstream lockB("./input_b.lock");
        inputIsReady = lockA.is_open() && lockB.is_open();
        std::cout << "Waiting for input files...\n";
        sleep(secondsToSleep);
    }
    std::cout << "Loading data...\n";

    std::cout << "Removing output files...\n";
    std::remove("./output_a.csv");
    std::remove("./output_a.lock");
    std::remove("./output_b.csv");
    std::remove("./output_b.lock");

    std::string lineA;
    std::string lineB;

    std::cout << "Parsing input files...\n";
    std::ifstream inputA("./input_a.csv");
    std::ifstream inputB("./input_b.csv");

    auto parseLine = [](float *data, std::string line)
    {
        size_t i = 0;
        std::string token;
        std::istringstream tokenStream(line);
        while (std::getline(tokenStream, token, ','))
        {
            data[i++] = std::stof(token);
        }
    };

    if (inputA.is_open() && inputB.is_open())
    {
        getline(inputA, lineA);
        parseLine(a, lineA);
        inputA.close();
        getline(inputB, lineB);
        parseLine(b, lineB);
        inputB.close();
    }
}

__host__ void saveOutputFile(float *results, size_t elementCount)
{
    std::cout << "Saving data...\n";
    std::ofstream outputA("./output_a.csv");
    std::ofstream outputB("./output_b.csv");

    for (size_t i = 0; i < elementCount; i++)
    {
        outputA << results[i] << (i == elementCount - 1 ? "" : ",");
        outputB << (0 - results[i]) << (i == elementCount - 1 ? "" : ",");
    }
    outputA << '\n';
    outputB << '\n';

    outputA.close();
    outputB.close();
}

#define EXPECTED_ARGC 3 // <element_count> <runs_to_execute> <seconds_to_sleep>

int main(int argc, char *argv[])
{
    if (argc != EXPECTED_ARGC + 1)
    {
        std::cout << "Usage: <element_count> <runs_to_execute> <seconds_to_sleep>\n";
        return EXIT_FAILURE;
    }

    size_t elementCount = std::stoul(argv[1]);
    size_t runsToExecute = std::stoul(argv[2]);
    secondsToSleep = std::stoul(argv[3]);

    float *a, *b, *results;
    hipMallocManaged(&a, elementCount * sizeof(float));
    hipMallocManaged(&b, elementCount * sizeof(float));
    hipMallocManaged(&results, elementCount * sizeof(float));

    for (size_t i = 0; i < runsToExecute; i++)
    {
        std::cout << "Run " << i + 1 << " of " << runsToExecute << '\n';

        loadInputFile(a, b, elementCount);
        executeKernel(a, b, results, elementCount);
        saveOutputFile(results, elementCount);

        remove("./input_a.lock");
        remove("./input_b.lock");

        auto signalOutputProcessed = [](const char *filename)
        {
            std::fstream lock;
            lock.open(filename, std::ios::out);
            lock.is_open();
            lock.close();
        };

        signalOutputProcessed("./output_a.lock");
        signalOutputProcessed("./output_b.lock");
    }

    hipFree(a);
    hipFree(b);
    hipFree(results);
    hipDeviceReset();

    std::cout << "Done!\n";

    return EXIT_SUCCESS;
}